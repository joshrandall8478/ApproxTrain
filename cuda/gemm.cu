#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <math.h>
#include <hip/hip_fp16.h>
#include "tensorflow/core/framework/types.h"
using namespace tensorflow;

#ifdef AMSIMULATOR
   #define MULTIPLY(a,b) AMsimulator((a), (b), mant_lut, mant_mask, a_shift, b_shift, mant_bitwidth);
   #include "AMsimulator.inl"
#else
   #define MULTIPLY(a,b) ((a)*(b));
#endif


#define TILE_DIM 16
/*
    Goal is to setup SEA emulation with (FP16,FP32), (BF16,FP32), (E4M3, FP32), (E5M2, FP32)
    Poritise (BF16, FP32), then somehow we generalise implementation
*/

#ifdef RTZ
    #define fp32_add(a,b) __fadd_rz((a), (b));
#else
    #define fp32_add(a,b) ((a)+(b));
#endif




/*
    The following is for half precision that accumulate in FP32
*/


// Convert FP32 to FP16
__device__ __half fp32_to_fp16(float a) {
    return __float2half(a);
}

// Convert FP16 back to FP32
__device__ float fp16_to_fp32(__half a) {
    return __half2float(a);
}


            

template <typename T>
__global__ void gemm(size_t m, size_t n, size_t k,
    const T *a, size_t lda, const T *b, size_t ldb,
   T *c, size_t ldc, hipTextureObject_t mant_lut,
   uint32_t mant_mask, uint8_t a_shift, uint8_t b_shift, uint8_t mant_bitwidth)
{
    T value(0);

    int Row = blockIdx.y*TILE_DIM + threadIdx.y;
    int Col = blockIdx.x*TILE_DIM + threadIdx.x;

    __shared__ T As[TILE_DIM][TILE_DIM];
    __shared__ T Bs[TILE_DIM][TILE_DIM];

    for (int i = 0; i < (TILE_DIM + k - 1)/TILE_DIM; ++i) {

        if (i*TILE_DIM + threadIdx.x < k && Row < m){
            As[threadIdx.y][threadIdx.x] = a[Row*lda + i*TILE_DIM + threadIdx.x];
        }
        else{
            As[threadIdx.y][threadIdx.x] = T(0);
        }

        if (i*TILE_DIM + threadIdx.y < k && Col < n){
            Bs[threadIdx.y][threadIdx.x] = b[(i*TILE_DIM + threadIdx.y)*ldb + Col];
        }
        else{
            Bs[threadIdx.y][threadIdx.x] = T(0);
        }

        __syncthreads();

        for (int n = 0; n < TILE_DIM; ++n){
            #ifdef FP16MUL
                // Convert to FP16, perform multiplication, convert back to FP32
                __half a_fp16 = fp32_to_fp16(As[threadIdx.y][n]);
                __half b_fp16 = fp32_to_fp16(Bs[n][threadIdx.x]);
                float mul_fp32 = fp16_to_fp32(__hmul(a_fp16, b_fp16));

                // Accumulate in FP32
                value = fp32_add(mul_fp32, value);
            #else
                T mul = MULTIPLY(As[threadIdx.y][n], Bs[n][threadIdx.x]);
                value = fp32_add(mul, value);
            #endif
        }

        __syncthreads();
    }

    if (Row < m && Col < n) {
        c[((blockIdx.y * blockDim.y  + threadIdx.y)*ldc) + (blockIdx.x * blockDim.x) + threadIdx.x] = value;
    }
}

template __global__ void gemm<float>(size_t m, size_t n, size_t k,
    const float *a, size_t lda, const float *b, size_t ldb,
   float *c, size_t ldc, hipTextureObject_t mant_lut,
   uint32_t mant_mask, uint8_t a_shift, uint8_t b_shift, uint8_t mant_bitwidth);
template __global__ void gemm<int32>(size_t m, size_t n, size_t k,
    const int32 *a, size_t lda, const int32 *b, size_t ldb,
   int32 *c, size_t ldc, hipTextureObject_t mant_lut,
   uint32_t mant_mask, uint8_t a_shift, uint8_t b_shift, uint8_t mant_bitwidth);
