#define EIGEN_USE_GPU
#include <vector>
#include <fstream>
#include <iostream>
#include <algorithm>
#include <iterator>
#include "error.cuh"
#include "approx_mul_lut.h"
using namespace tensorflow;

// Constructor Implementation
template<>
approx_mul_lut<Eigen::GpuDevice>::approx_mul_lut(tensorflow::OpKernelConstruction* context)
    : approx_mul_lut_base(context) {
    if (!lut_)
        return;
    if (fp8_) {
        // Handle combined FP8 LUT
        gpuErrchk(hipMalloc(&mant_mul_lut_cuda_fp32_,
                             mant_mul_lut_fp32_.size() * sizeof(float)));
        gpuErrchk(hipMemcpy(mant_mul_lut_cuda_fp32_, mant_mul_lut_fp32_.data(),
                             mant_mul_lut_fp32_.size() * sizeof(float),
                             hipMemcpyHostToDevice));

        // Create texture object for combined FP8 LUT
        hipResourceDesc res_desc;
        memset(&res_desc, 0, sizeof(hipResourceDesc));
        res_desc.resType = hipResourceTypeLinear;
        res_desc.res.linear.devPtr = mant_mul_lut_cuda_fp32_;
        res_desc.res.linear.desc.f = hipChannelFormatKindFloat;
        res_desc.res.linear.desc.x = 32;
        res_desc.res.linear.sizeInBytes =
            mant_mul_lut_fp32_.size() * sizeof(float);

        hipTextureDesc tex_desc;
        memset(&tex_desc, 0, sizeof(hipTextureDesc));
        tex_desc.readMode = hipReadModeElementType;

        gpuErrchk(hipCreateTextureObject(&mant_mul_lut_text_,
                                          &res_desc,
                                          &tex_desc, nullptr));
    } else {
        // Handle 8-bit LUT
        gpuErrchk(hipMalloc(&mant_mul_lut_cuda_uint8_,
                             mant_mul_lut_uint8_.size() * sizeof(uint8_t)));
        gpuErrchk(hipMemcpy(mant_mul_lut_cuda_uint8_, mant_mul_lut_uint8_.data(),
                             mant_mul_lut_uint8_.size() * sizeof(uint8_t),
                             hipMemcpyHostToDevice));

        // Create texture object for 8-bit LUT
        hipResourceDesc res_desc;
        memset(&res_desc, 0, sizeof(hipResourceDesc));
        res_desc.resType = hipResourceTypeLinear;
        res_desc.res.linear.devPtr = mant_mul_lut_cuda_uint8_;
        res_desc.res.linear.desc.f = hipChannelFormatKindUnsigned;
        res_desc.res.linear.desc.x = 8;
        res_desc.res.linear.sizeInBytes =
            mant_mul_lut_uint8_.size() * sizeof(uint8_t);

        hipTextureDesc tex_desc;
        memset(&tex_desc, 0, sizeof(hipTextureDesc));
        tex_desc.readMode = hipReadModeElementType;

        gpuErrchk(hipCreateTextureObject(&mant_mul_lut_text_,
                                          &res_desc,
                                          &tex_desc, nullptr));
    }
}

// Destructor Implementation
template<>
approx_mul_lut<Eigen::GpuDevice>::~approx_mul_lut() {
    // check if lut is enabled
    if (!lut_)
        return;
    hipDestroyTextureObject(mant_mul_lut_text_);
    if (fp8_) {
        hipFree(mant_mul_lut_cuda_fp32_);
    } else {
        hipFree(mant_mul_lut_cuda_uint8_);
    }
}
