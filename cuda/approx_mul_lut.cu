#define EIGEN_USE_GPU
#include <vector>
#include <fstream>
#include <iostream>
#include <algorithm>
#include <iterator>
#include "error.cuh"
#include "approx_mul_lut.h"
using namespace tensorflow;

template<>
class approx_mul_lut<Eigen::GpuDevice> : public approx_mul_lut_base {
    public:
        explicit approx_mul_lut(tensorflow::OpKernelConstruction* context);
        ~approx_mul_lut();
        auto get_mant_mul_lut_() -> void* {
            if (fp8_) {
                return mant_mul_lut_cuda_fp32_;
            } else {
                return mant_mul_lut_cuda_uint8_;
            }
        }

        auto get_mant_mul_lut_text_() -> hipTextureObject_t& {
            return mant_mul_lut_text_;
        }
};


approx_mul_lut<Eigen::GpuDevice>::approx_mul_lut(OpKernelConstruction* context)
    : approx_mul_lut_base(context) {
    if (fp8_) {
        // Handle 32-bit float LUT
        // Allocate CUDA memory for the FP32 LUT
        gpuErrchk(hipMalloc(&mant_mul_lut_cuda_fp32_,
                             mant_mul_lut_fp32_.size() * sizeof(float)));
        gpuErrchk(hipMemcpy(mant_mul_lut_cuda_fp32_, mant_mul_lut_fp32_.data(),
                             mant_mul_lut_fp32_.size() * sizeof(float),
                             hipMemcpyHostToDevice));

        // Create texture object for FP32 LUT
        hipResourceDesc mant_mul_lut_res_desc;
        memset(&mant_mul_lut_res_desc, 0, sizeof(hipResourceDesc));
        mant_mul_lut_res_desc.resType = hipResourceTypeLinear;
        mant_mul_lut_res_desc.res.linear.devPtr = mant_mul_lut_cuda_fp32_;
        mant_mul_lut_res_desc.res.linear.desc.f = hipChannelFormatKindFloat;
        mant_mul_lut_res_desc.res.linear.desc.x = 32;
        mant_mul_lut_res_desc.res.linear.sizeInBytes =
            mant_mul_lut_fp32_.size() * sizeof(float);

        hipTextureDesc mant_mul_text_desc;
        memset(&mant_mul_text_desc, 0, sizeof(hipTextureDesc));
        mant_mul_text_desc.readMode = hipReadModeElementType;

        gpuErrchk(hipCreateTextureObject(&mant_mul_lut_text_,
                                          &mant_mul_lut_res_desc,
                                          &mant_mul_text_desc, nullptr));
    } else {
        // Handle 8-bit LUT
        // Allocate CUDA memory for the uint8 LUT
        gpuErrchk(hipMalloc(&mant_mul_lut_cuda_uint8_,
                             mant_mul_lut_uint8_.size() * sizeof(uint8_t)));
        gpuErrchk(hipMemcpy(mant_mul_lut_cuda_uint8_, mant_mul_lut_uint8_.data(),
                             mant_mul_lut_uint8_.size() * sizeof(uint8_t),
                             hipMemcpyHostToDevice));

        // Create texture object for uint8 LUT
        hipResourceDesc mant_mul_lut_res_desc;
        memset(&mant_mul_lut_res_desc, 0, sizeof(hipResourceDesc));
        mant_mul_lut_res_desc.resType = hipResourceTypeLinear;
        mant_mul_lut_res_desc.res.linear.devPtr = mant_mul_lut_cuda_uint8_;
        mant_mul_lut_res_desc.res.linear.desc.f = hipChannelFormatKindUnsigned;
        mant_mul_lut_res_desc.res.linear.desc.x = 8;
        mant_mul_lut_res_desc.res.linear.sizeInBytes =
            mant_mul_lut_uint8_.size() * sizeof(uint8_t);

        hipTextureDesc mant_mul_text_desc;
        memset(&mant_mul_text_desc, 0, sizeof(hipTextureDesc));
        mant_mul_text_desc.readMode = hipReadModeElementType;

        gpuErrchk(hipCreateTextureObject(&mant_mul_lut_text_,
                                          &mant_mul_lut_res_desc,
                                          &mant_mul_text_desc, nullptr));
    }
}


approx_mul_lut<Eigen::GpuDevice>::~approx_mul_lut() {
    hipDestroyTextureObject(mant_mul_lut_text_);
    if (fp8_) {
        hipFree(mant_mul_lut_cuda_fp32_);
    } else {
        hipFree(mant_mul_lut_cuda_uint8_);
    }
}